// #include <cstdlib>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <cmath>
#include <hipblas.h>


__host__ inline void randn_matrices (float* x, float* y, int x_height, int y_width, int num_prod) {
    std::random_device rnd;
    std::default_random_engine eng(rnd());
    std::uniform_real_distribution<float> distr(-1, 1);

    for (int i=0; i < x_height * num_prod; i++) {
        x[i] = distr(eng);
    }

    for (int i=0; i < num_prod * y_width; i++) {
        y[i] = distr(eng);
    }
}

inline void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int n, const int k) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}


void test(int size, int iterations) {
    // Allocate 3 arrays on CPU
    int x_height = size;
    int num_prod = size;
    int y_width = size;

    float *x = (float *)malloc(x_height * num_prod * sizeof(float));
    float *y = (float *)malloc(num_prod * y_width * sizeof(float));
    float *out = (float *)malloc(num_prod * num_prod * sizeof(float));
    randn_matrices(x, y, x_height, y_width, num_prod);

    double d_sec_memcpy_cuda = 0;
    double d_sec_kernel_cuda = 0;
    double d_sec_total_cuda = 0;

    for (int num_iter=0; num_iter<(iterations+1); num_iter++) {
        hipEvent_t start, memh2d_stop, kernel_stop, memd2h_stop;
        hipEventCreate(&start);
        hipEventCreate(&memh2d_stop);
        hipEventCreate(&kernel_stop);
        hipEventCreate(&memd2h_stop);
        hipEventRecord(start);

        // Allocate 3 arrays on GPU
        float *d_x, *d_y, *d_out;
        hipMalloc(&d_x, x_height * num_prod * sizeof(float));
        hipMalloc(&d_y, num_prod * y_width * sizeof(float));
        hipMalloc(&d_out, num_prod * num_prod * sizeof(float));
        
        // Optionally we can copy the data back on CPU and print the arrays
        hipMemcpy(x, d_x, x_height * num_prod  * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(y, d_y, num_prod * y_width * sizeof(float), hipMemcpyDeviceToHost);

        hipEventRecord(memh2d_stop);
        hipEventSynchronize(memh2d_stop);

        // Multiply A and B on GPU
        gpu_blas_mmul(d_x, d_y, d_out, x_height, y_width, num_prod);

        hipEventRecord(kernel_stop);
        hipEventSynchronize(kernel_stop);

        // Copy (and print) the result on host memory
        hipMemcpy(out, d_out, num_prod * num_prod * sizeof(float), hipMemcpyDeviceToHost);

        hipEventRecord(memd2h_stop);
        hipEventSynchronize(memd2h_stop);

        if (num_iter != 0) {
            float milisec = 0.0;
            hipEventElapsedTime(&milisec, start, memd2h_stop);
            d_sec_total_cuda += milisec / 1000;
            hipEventElapsedTime(&milisec, start, memh2d_stop);
            d_sec_memcpy_cuda += milisec / 1000;
            hipEventElapsedTime(&milisec, memh2d_stop, kernel_stop);
            d_sec_kernel_cuda += milisec / 1000;
        }

        hipEventDestroy(start);
        hipEventDestroy(memh2d_stop);
        hipEventDestroy(kernel_stop);
        hipEventDestroy(memd2h_stop);

        //Free GPU memory
        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_out);
    }

    hipDeviceReset();

    d_sec_memcpy_cuda /= iterations;
    d_sec_kernel_cuda /= iterations;
    d_sec_total_cuda /= iterations;

    printf("行列サイズ=%d\n", size);
    printf("計算結果=%f\n", out[x_height * y_width - 1]);
    printf("処理時間 : メモリ移動 = %lf\n", d_sec_memcpy_cuda);
    printf("処理時間 : カーネル実行 =%lf\n", d_sec_kernel_cuda);
    printf("処理時間 : トータル =%lf\n", d_sec_total_cuda);

    // Free CPU memory
    free(x);
    free(y);
    free(out);
}


int main() {
    int iterations = 10;
    int sizes[3] = {256, 1024, 4096};

    for (auto& size : sizes) {
        test(size, iterations);
    }

    return 0;
}