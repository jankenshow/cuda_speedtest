
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <time.h>
#include <sys/time.h>
// #include <chrono>
#include <random>
// #include <iomanip>
#include <cmath>


__global__ void matmul(float* out, float* x, float* y, int height, int width, int num_prod) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < width && idy < height) {
        float tmp = 0.0;
        for (int k=0; k < num_prod; k++) {
            // int x_id = k + idy * num_prod;
            // int y_id = idx + k * num_prod;
            // tmp += x[x_id] * y[y_id];
            tmp += x[k + idy * num_prod] * y[idx + k * num_prod];
        }
        out[idx + idy * width] = tmp;
    }
}

__host__ inline void randn_matrices(float* x, float* y, int x_height, int y_width, int num_prod) {
    std::random_device rnd;
    std::default_random_engine eng(rnd());
    std::uniform_real_distribution<float> distr(-1, 1);

    for (int i=0; i < x_height * num_prod; i++) {
        x[i] = distr(eng);
    }

    for (int i=0; i < num_prod * y_width; i++) {
        y[i] = distr(eng);
    }
}

void test(int size, int iterations) {
    int x_height = size;
    int num_prod = size;
    int y_width = size;

    int thread_size = 32;

    float* x = new float[x_height * num_prod];
    float* y = new float[num_prod * y_width];
    float* out = new float[x_height * y_width];
    randn_matrices(x, y, x_height, y_width, num_prod);

    struct timespec start_time, memcpy_end_time, kernel_end_time, total_end_time;
    unsigned int sec;
    int nsec;
    double d_sec_memcpy = 0;
    double d_sec_kernel = 0;
    double d_sec_total = 0;

    double d_sec_memcpy_cuda = 0;
    double d_sec_kernel_cuda = 0;
    double d_sec_total_cuda = 0;

    for (int num_iter=0; num_iter<(iterations+1); num_iter++) {

        // 時間計測
        clock_gettime(CLOCK_REALTIME, &start_time);

        hipEvent_t start, memh2d_stop, kernel_stop, memd2h_stop;
        hipEventCreate(&start);
        hipEventCreate(&memh2d_stop);
        hipEventCreate(&kernel_stop);
        hipEventCreate(&memd2h_stop);
        hipEventRecord(start);


        // メモリ移動 host -> device
        float* d_x = nullptr;
        float* d_y = nullptr;
        float* d_out = nullptr;

        hipMalloc((void **)&d_x, sizeof(float) * x_height * num_prod);
        hipMalloc((void **)&d_y, sizeof(float) * num_prod * y_width);
        hipMalloc((void **)&d_out, sizeof(float) * x_height * y_width);

        hipMemcpy(d_x, x, sizeof(float) * x_height * num_prod, hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, sizeof(float) * num_prod * y_width, hipMemcpyHostToDevice);


        // 時間計測
        if (num_iter != 0) {
            hipDeviceSynchronize();
            clock_gettime(CLOCK_REALTIME, &memcpy_end_time);
            sec = memcpy_end_time.tv_sec - start_time.tv_sec;
            nsec = memcpy_end_time.tv_nsec - start_time.tv_nsec;
            d_sec_memcpy += (double)sec * 1000 + (double)nsec / (1000 * 1000);
        }

        hipEventRecord(memh2d_stop);
        hipEventSynchronize(memh2d_stop);


        // kernel実行
        dim3 grid(std::ceil(x_height / thread_size), std::ceil(y_width / thread_size), 1);
        dim3 threads(thread_size, thread_size, 1);

        matmul<<<grid, threads>>>(d_out, d_x, d_y, x_height, y_width, num_prod);


        // 時間計測
        if (num_iter != 0) {
            hipDeviceSynchronize();
            clock_gettime(CLOCK_REALTIME, &kernel_end_time);
            sec = kernel_end_time.tv_sec - memcpy_end_time.tv_sec;
            nsec = kernel_end_time.tv_nsec - memcpy_end_time.tv_nsec;
            d_sec_kernel += (double)sec * 1000 + (double)nsec / (1000 * 1000);
        }

        hipEventRecord(kernel_stop);
        hipEventSynchronize(kernel_stop);


        // メモリ移動 device -> host
        hipMemcpy(out, d_out, sizeof(float) * x_height * y_width, hipMemcpyDeviceToHost);


        // 時間計測
        if (num_iter != 0) {
            hipDeviceSynchronize();
            clock_gettime(CLOCK_REALTIME, &total_end_time);
            sec = total_end_time.tv_sec - start_time.tv_sec;
            nsec = total_end_time.tv_nsec - start_time.tv_nsec;
            d_sec_total += (double)sec * 1000 + (double)nsec / (1000 * 1000);
        }

        hipEventRecord(memd2h_stop);
        hipEventSynchronize(memd2h_stop);

        if (num_iter != 0) {
            float milisec = 0.0;
            hipEventElapsedTime(&milisec, start, memd2h_stop);
            d_sec_total_cuda += milisec;
            hipEventElapsedTime(&milisec, start, memh2d_stop);
            d_sec_memcpy_cuda += milisec;
            hipEventElapsedTime(&milisec, memh2d_stop, kernel_stop);
            d_sec_kernel_cuda += milisec;
        }

        // 時間計測後処理
        hipEventDestroy(start);
        hipEventDestroy(memh2d_stop);
        hipEventDestroy(kernel_stop);
        hipEventDestroy(memd2h_stop);


        // kernel実行後処理
        // printf("計算結果 = %f\n", out[x_height * y_width - 1]);

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_out);
    }

    hipDeviceReset();

    d_sec_memcpy /= iterations;
    d_sec_kernel /= iterations;
    d_sec_total /= iterations;

    d_sec_memcpy_cuda /= iterations;
    d_sec_kernel_cuda /= iterations;
    d_sec_total_cuda /= iterations;

    printf("行列サイズ=%d\n", size);
    printf("計算結果=%f\n", out[x_height * y_width - 1]);
    printf("処理時間 : メモリ移動 = %lf vs %lf\n", d_sec_memcpy, d_sec_memcpy_cuda);
    printf("処理時間 : カーネル実行 =%lf vs %lf\n", d_sec_kernel, d_sec_kernel_cuda);
    printf("処理時間 : トータル =%lf vs %lf\n", d_sec_total, d_sec_total_cuda);

    delete[] x;
    delete[] y;
    delete[] out;
}


int main() {
    int iterations = 10;
    int sizes[3] = {256, 1024, 4096};

    for (auto& size : sizes) {
        test(size, iterations);
    }

    return 0;
}